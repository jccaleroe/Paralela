#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

int N, *D_N, threads_per_block, blocks_num, threads_num, *d_threads_num;
float *A, *B, *C, *D_A, *D_B, *D_C, max_num = (float)(RAND_MAX);

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__global__ void multiply( float *A, float *B, float *C, int N, int threads_num ){

 	float tmp;
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
 	int i, j, k, pos, cell_per_thread = N * N / threads_num;

	int a = cell_per_thread * index, b = cell_per_thread * (index + 1);

	if (index == threads_num - 1)
        b = N * N;

    for( pos = a; pos < b; pos++ ){
        i = pos / N;
        j = pos - i*N;
        tmp = 0;
        for(k = 0; k < N; k++)
            tmp += A[i*N + k] * B[k*N + j];
        C[i*N + j] = tmp;
    }
}

int main(int argc, char *agcv){
	if(argc != 4){
        printf("Please enter matrix size N\n");
        scanf("%d", &N);
        printf("Please enter number of blocks\n");
        scanf("%d", &blocks_num);
		printf("Please enter number of threads per block\n");
        scanf("%d", &threads_num);
    }
    else{
        N = atoi(argv[1]);
        blocks_num = atoi(argv[2]);
		threads_num = atoi(argv[3]);
    }
	
	int size = N * N * sizeof(float), i, j;

	A = (float *) malloc ( size );
	B = (float *) malloc ( size );
	C = (float *) malloc ( size );

//    srand(time(NULL));
//    for( i = 0; i < N; i++){
//        for( j = 0; j < N; j++){
//            A[i*N + j] = max_num / (float) (rand());
//            B[i*N + j] = max_num / (float) (rand());
//        }
//    }

//    test case
    for( i = 0; i < N; i++){
        for( j = 0; j < N; j++){
            A[i*N + j] = i*N+j;
            B[i*N + j] = i*N+j;
        }
    }

	threads_per_block = threads_num / blocks_num;

    GPUerrchk( hipMalloc ( (void **) &D_A, size ) );
	GPUerrchk( hipMemcpy ( D_A, A, size, hipMemcpyHostToDevice );

    GPUerrchk( hipMalloc ( (void **) &D_B, size ) );
	GPUerrchk( hipMemcpy ( D_B, B, size, hipMemcpyHostToDevice );

    GPUerrchk( hipMalloc ( (void **) &D_C, size ) );
	GPUerrchk( hipMemcpy ( D_C, C, size, hipMemcpyHostToDevice );

    GPUerrchk( hipMalloc ( (void **) &D_N, sizeof(int) ) );
	GPUerrchk( hipMemcpy ( D_N, &N, sizeof(int), hipMemcpyHostToDevice );

    GPUerrchk( hipMalloc ( (void **) &D_threads_num, sizeof(int) ) );
	GPUerrchk( hipMemcpy ( D_threads_num, &threads_num, sizeof(int), hipMemcpyHostToDevice );

    multiply<<<blocks_num, threads_per_block>>>(D_A, D_B, D_C, D_N, D_threads_num);
	
	GPUerrchk( cudaPeeKAtLastError() );
	
	GPUerrchk( hipMemcpy(C, D_C, size, hipMemcpyDeviceToHost) );

//  print matrix
    for( i = 0; i < N; i++){
        for( j = 0; j < N; j++)
          printf("%0.1f ", C[i*N +j]);
        printf("\n");
    }

    free(A); free(B); free(C);
	GPUerrchk ( hipDeviceReset() );
}
