#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define GPUERRCHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int N, threads_per_block, blocks_num, threads_num;
float *A, *B, *C, *D_A, *D_B, *D_C, max_num = (float)(RAND_MAX);

__global__ void multiply( float *A2, float *B2, float *C, int N, int threads_num ){
	__shared__ float *A;
	__shared__ float *B;
	A = A2;	B = B2;

 	float tmp;
 	int k, pos;

	int a = N * N * (blockDim.x * blockIdx.x + threadIdx.x) / threads_num, b;

	if ( blockDim.x * blockIdx.x + threadIdx.x == threads_num - 1)
        b = N * N;
	else
		b = N * N * ( blockDim.x * blockIdx.x + threadIdx.x + 1) / threads_num;

    for( pos = a; pos < b; pos++ ){
        tmp = 0;
        for( k = 0; k < N; k++ )
            tmp += A[ N * (pos / N ) + k ] * B[ k * N + pos - ( pos / N) * N];
        C[ pos ] = tmp;
    }
}

int main(int argc, char *argv[]){
	if(argc != 4){
        printf("Please enter matrix size N\n");
        scanf("%d", &N);
        printf("Please enter number of blocks\n");
        scanf("%d", &blocks_num);
		printf("Please enter number of threads\n");
        scanf("%d", &threads_per_block);
    }
    else{
        N = atoi(argv[1]);
        blocks_num = atoi(argv[2]);
		threads_per_block = atoi(argv[3]);
    }
	
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventRecord(start, 0);

	int size = N * N * sizeof(float), i, j;

	A = (float *) malloc ( size );
	B = (float *) malloc ( size );
	C = (float *) malloc ( size );

    srand(time(NULL));
    for( i = 0; i < N; i++){
        for( j = 0; j < N; j++){
            A[i*N + j] = max_num / (float) (rand());
            B[i*N + j] = max_num / (float) (rand());
        }
    }

//   test case
//    for( i = 0; i < N; i++){
//        for( j = 0; j < N; j++){
//            A[i*N + j] = i*N+j;
//            B[i*N + j] = i*N+j;
//        }
//    }

	threads_num	= threads_per_block * blocks_num;

    GPUERRCHK( hipMalloc ( (void **) &D_A, size ) );
	GPUERRCHK( hipMemcpy ( D_A, A, size, hipMemcpyHostToDevice ) );

    GPUERRCHK( hipMalloc ( (void **) &D_B, size ) );
	GPUERRCHK( hipMemcpy ( D_B, B, size, hipMemcpyHostToDevice ) );

    GPUERRCHK( hipMalloc ( (void **) &D_C, size ) );

    multiply<<<blocks_num, threads_per_block>>>(D_A, D_B, D_C, N, threads_num);
	
	GPUERRCHK( hipPeekAtLastError() );
	
	GPUERRCHK( hipMemcpy(C, D_C, size, hipMemcpyDeviceToHost) );

//    print matrix
//    for( i = 0; i < N; i++){
//        for( j = 0; j < N; j++)
//          printf("%0.1f ", C[i*N +j]);
//        printf("\n");
//    }

    free(A); free(B); free(C);

	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Elapsed time: %f ms\n", elapsedTime);

	GPUERRCHK ( hipDeviceReset() );
}
